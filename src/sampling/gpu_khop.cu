#include "hip/hip_runtime.h"
#include "graph_gpu.h"
#include <bits/stdc++.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
using namespace std;
using namespace cooperative_groups;

const int BLOCK_DIM = 32;

inline double seconds() {
    struct timeval tp;
    struct timezone tzp;
    gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}


__device__ __inline__ int next(GraphGPU &g, int transit, int idx, hiprandState state) {
    // return transit * 10 + idx;
    int n = g.getOutDegree(transit);
    if (n == 0) {
        return transit;
    }
    int chosen = (int)(ceil(hiprand_uniform(&state) * n) - 1);
    return g.N(transit, chosen);
}

__global__ void baseline(GraphGPU g, int* result, int steps, int* sample_size, int cur_num, int total_threads, int seed) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix >= total_threads) {
        return;
    }
    int d, r, low, high;
    int prev_begin = 0;
    int begin = cur_num;
    int transit, idx;
    hiprandState ix_state;
    hiprand_init(seed, ix, 0, &ix_state);
    grid_group grid = this_grid();
    for (int i = 0; i < steps; i++) {
        cur_num *= sample_size[i];
        d = cur_num / total_threads; 
        r = cur_num % total_threads;
        if (ix < r) {
            low = ix * (d + 1);
            high = (ix + 1) * (d + 1);
        } else {
            low = r + ix * d;
            high = r + ix * d + d;
        }
        for (int j = low; j < high; j++) {
            idx = j % sample_size[i];
            transit = result[j / sample_size[i] + prev_begin];
            result[begin + j] = next(g, transit, idx, ix_state);
        }
        prev_begin = begin;
        begin += cur_num;        
        grid.sync();
        
    }
} 
// 2 * 2 *3

// 1, 2
// 11, 12, 21, 22
// 111, 112, 113, 121, 122, 123, 211, 212, 213, 221, 222, 223
// 40000 * 25 * 10 + 40000 * 25 + 40000
double khop_sample(Graph &g, vector<int>& initial, int steps, int* sample_size, int total_num, int* result, int pdeg=128, int seed=0) {
    GraphGPU gpu_g (g);
    int cur_num = initial.size();
    int *d_result, *d_sample_size;
    int total_threads = pdeg;
    dim3 block(BLOCK_DIM);
    dim3 grid((total_threads + block.x - 1) / block.x);
    double iStart, iElaps, oStart, oElaps = 0;
    for (int i = 0; i < cur_num; i++) {
        result[i] = initial[i];
    }
    
    oStart = seconds();
    CUDA_SAFE_CALL(hipMalloc((void **)&d_result, total_num * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_sample_size, steps * sizeof(int)));

    CUDA_SAFE_CALL(hipMemcpy(d_result, result, cur_num * sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_sample_size, sample_size, steps * sizeof(int), hipMemcpyHostToDevice));

    void *kernel_args[] = {&gpu_g, &d_result, &steps, &d_sample_size, &cur_num, &total_threads, &seed};
    oElaps += seconds() - oStart;
    iStart = seconds();
    hipLaunchCooperativeKernel((void*)(baseline), grid, block, kernel_args);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iElaps = seconds() - iStart;

    oStart = seconds();
    CUDA_SAFE_CALL(hipMemcpy(result, d_result, total_num * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(d_result));
    CUDA_SAFE_CALL(hipFree(d_sample_size));
    oElaps += seconds() - oStart;

    cout << "Time elapsed for allocating and copying " << oElaps << " sec\n\n";

    return iElaps;
}

// cpu:1.41311e-3
// gpu:
// kernel: 4.19617e-5
// allocating and copying others: 4.68731e-4
// allocating graph: 0.15959
// copying graph: 8.2e-5